//
//  stencil main program
//  using CUDA to do parallel computing of stencil
//  do t repeated run
//  do it for n = 500, 1000, 20000 and  t = 10
//  Created by Ethan Zhang on 11/8/18.
//  Copyright © 2018 Ethan Zhang. All rights reserved.
//


#include <iostream>
#include <algorithm>
#include "stdio.h"
#include "cmath"
#include <float.h>
#include "hip/hip_runtime.h"
#include <ctime>

#define THREADS_PER_DIM 25
// #define TASKS_PER_THREADS 50
// #define BLOCKS 32
// #define N 1000*1000
// #define RADIUS  1001
// #define TASKS 
using namespace std;

__device__
double get2ndMin(double *candidates){
    double first, second;
    first = second = DBL_MAX;
    for(int k =0; k<4; k++){
        if(candidates[k]<=first){
            second = first;
            first = candidates[k];
        }
        else if (candidates[k] <= second && candidates[k] >= first){
            second = candidates[k];}
    }
    return second;
}


__global__ 
void calc(int n, double *dA, double *prev_dA){

    int j = threadIdx.y + blockIdx.y * blockDim.y; 
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(i ==0 || i ==n-1 || j ==0 || j ==n-1){
        dA[i*n+j] = prev_dA[i*n+j];
    }else{
        // tmp[lindex_x-1][lindex_y-1] = A[i-1][j-1]
        double candidates[] = {prev_dA[(i+1)*n+(j+1)], prev_dA[(i+1)*n+(j-1)],prev_dA[(i-1)*n+(j-1)],prev_dA[(i-1)*n+(j+1)]};
        dA[i*n+j] = prev_dA[i*n+j] + get2ndMin(candidates);
    }
    __syncthreads();
    // printf("exec. in block%d, threads%d, i%d, j%d, \n", blockIdx.x, threadIdx.x, i, j);
}

//parent node
// __global__ void stencil(double *dA,int n){

//     calc<<<BLOCKS, THREADS_PER_DIM>>>(n, dA); 
//     __syncthreads();
//     printf("exec. in parent node\n");
// }

__global__ void reduce(double *g_idata, double *g_odata) {
    extern __shared__ double sdata[];
    // each thread loads one element from global to shared mem
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x *blockDim.x +threadIdx.y;

    unsigned int i = (blockIdx.x *blockDim.x + blockIdx.y)*(blockDim.x*2) + tid; // global index, threads in previous blocks and 
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=1;s<blockDim.x *blockDim.y; s++) {
        if (tid < s) {
            sdata[tid] += sdata[tid + 1];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x *blockDim.x + blockIdx.y] = sdata[0];
}




__global__
void verification(double *A, int n){
    double v1, v2,v3;
    v1 = 0.0;
    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++){
            v1 += A[i*n+j];
        }
    }

    int fl = floor((double)n/2);
    v2 = A[fl*n+fl];
    v3 = A[37*n+47];
    // __syncthreads();
    A[0] = v1;
    A[1] = v2;
    A[2] = v3; 
}

double verisum_all(int n, double *A){
    double sum=0.0;
    for(int i = 0; i<n; i++){
        for(int j=0; j<n; j++){
            sum += A[i*n+j];
        }
    }
    return sum;
}

double value_half(int n, double *A){
    int fl = floor((double)n/2);
    double result  = A[fl*n+ fl];
    return result;
}

double value_37_47(int n, double *A){
    double result =A[37*n+47];
    return result;
}

int main(int argc, char** argv) {
    // initialize below
    int n = atoi(argv[1]);

    int N  = n*n;
    printf("size N%d\n",N);

    // initialize below
    // 1d stencil

    double *array;
    double *sum;
    int step = n/THREADS_PER_DIM; 
    int size = (N) * sizeof(double);

    int g_size = (step*step) * sizeof(double); 
    array =(double *)malloc(size);
    sum = (double *)malloc(g_size);

    for(int i =0; i<n;i++){
        for(int j =0; j<n; j++){
            array[i*n+j] = pow(1+cos(2*i)+sin(j),2);
        }
    }

    for(int i=0; i<step; i++){
        for(int j=0; j<step; j++){
            sum[i*step+j] =0.0;
        }
    }


    //verify initialization results
    double verisum_1 = verisum_all(n, array);
    double half_value_1 = value_half(n, array);
    double spec_1  = value_37_47(n, array);
    
    //print result
    printf("init verisum all %f\n", verisum_1);
    printf("init verification n/2 %f\n", half_value_1);
    printf("init verification A[37][47] %f\n", spec_1);

    double *dA;
    double *prev_dA;
    double *g_out;
    
    // allocate memory on device
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&prev_dA, size);
    hipMalloc((void **)&g_out, g_size);

    // Copy inputs to device
    hipMemcpy(dA, array, size, hipMemcpyHostToDevice);
    hipMemcpy(prev_dA, array, size, hipMemcpyHostToDevice);
    hipMemcpy(g_out, sum, g_size, hipMemcpyHostToDevice);

    //launch kernal on device
    int t  = 10;
    dim3 dimBlock(THREADS_PER_DIM, THREADS_PER_DIM);
    dim3 dimGrid(n/THREADS_PER_DIM, n/ THREADS_PER_DIM);
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // double v1 =0.0; 
    
    hipEventRecord(start, 0);

    for(int episode =0; episode<t; episode++){
        // printf("loop %d\n", episode );
        calc<<<dimGrid, dimBlock>>>(n, dA, prev_dA);
        hipDeviceSynchronize();

        double *tem_a = dA;
        dA = prev_dA;
        prev_dA = tem_a;  
    }
    

    // verification<<<1,1>>>(prev_dA,n); //  para1 verification 
    
    hipMemcpy(array,prev_dA, size, hipMemcpyDeviceToHost);
    
    reduce<<<dimGrid,dimBlock, dimBlock.x *dimBlock.y *sizeof(double)>>>(prev_dA,g_out); //better verification
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy(sum,g_out, g_size, hipMemcpyDeviceToHost);
    double verisum=0;
    for(int i=0; i<step*step; i++){
        verisum += sum[i];
    }
        // print result
    printf ("Time for the kernel: %f ms\n", time);
    printf("verisum all %f\n", verisum);
    printf("verification n/2 %f\n", array[1]);
    printf("verification A[37][47] %f\n", array[2]);





    //free memory
    free(array);
    // free(sum);
    hipFree(dA);
    hipFree(prev_dA);
    // cudaFree(g_out);

    return 0;
}
