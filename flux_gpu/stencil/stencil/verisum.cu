//
//  stencil main program
//  using CUDA to do parallel computing of stencil
//  do t repeated run
//  do it for n = 500, 1000, 20000 and  t = 10
//  Created by Ethan Zhang on 11/8/18.
//  Copyright © 2018 Ethan Zhang. All rights reserved.
//


#include <iostream>
#include <algorithm>
#include "stdio.h"
#include "cmath"
#include <float.h>
#include "hip/hip_runtime.h"
#include <ctime>

#define THREADS_PER_DIM 25
// #define TASKS_PER_THREADS 50
// #define BLOCKS 32
// #define N 1000*1000
// #define RADIUS  1001
// #define TASKS 
using namespace std;

__device__
double get2ndMin(double *candidates){
    double first, second;
    first = second = DBL_MAX;
    for(int k =0; k<4; k++){
        if(candidates[k]<=first){
            second = first;
            first = candidates[k];
        }
        else if (candidates[k] <= second && candidates[k] >= first){
            second = candidates[k];}
    }
    return second;
}


__global__ 
void calc(int n, double *dA, double *prev_dA){

    int j = threadIdx.y + blockIdx.y * blockDim.y; 
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(i ==0 || i ==n-1 || j ==0 || j ==n-1){
        dA[i*n+j] = prev_dA[i*n+j];
    }else{
        // tmp[lindex_x-1][lindex_y-1] = A[i-1][j-1]
        double candidates[] = {prev_dA[(i+1)*n+(j+1)], prev_dA[(i+1)*n+(j-1)],prev_dA[(i-1)*n+(j-1)],prev_dA[(i-1)*n+(j+1)]};
        dA[i*n+j] = prev_dA[i*n+j] + get2ndMin(candidates);
    }
    __syncthreads();
    // printf("exec. in block%d, threads%d, i%d, j%d, \n", blockIdx.x, threadIdx.x, i, j);
}

//parent node
// __global__ void stencil(double *dA,int n){

//     calc<<<BLOCKS, THREADS_PER_DIM>>>(n, dA); 
//     __syncthreads();
//     printf("exec. in parent node\n");
// }


__global__ void reduce(double *g_idata, int n, int step, double *g_odata) {
    extern __shared__ double sdata[];
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x*blockDim.x+threadIdx.y;

    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
    sdata[tid] = g_idata[i*n+j];
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x*blockDim.y; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x *step+blockIdx.y] = sdata[0]; //thread 0 get the 
}




// __global__
// void verification(double *A, int n){
//     int j = threadIdx.y + blockIdx.y * blockDim.y; 
//     int i = threadIdx.x + blockIdx.x * blockDim.x;

//     sum += A[i*n+j];
//     __syncthreads();

//     // A[0] = sum;
//     double v2, v3; 
//     int fl = floor((double)n/2);
//     v2 = A[fl*n+fl];
//     v3 = A[37*n+47];
//     // __syncthreads()
//     A[1] = v2;
//     A[2] = v3; 
// }


double verisum_all(int n, double *A){
    double sum=0.0;
    for(int i = 0; i<n; i++){
        for(int j=0; j<n; j++){
            sum += A[i*n+j];
        }
    }
    return sum;
}

double value_half(int n, double *A){
    int fl = floor((double)n/2);
    double result  = A[fl*n+ fl];
    return result;
}

double value_37_47(int n, double *A){
    double result =A[37*n+47];
    return result;
}

int main(int argc, char** argv) {
    // initialize below
    int n = atoi(argv[1]);

    int N  = n*n;
    printf("size N%d\n",N);

    // initialize below
    // 1d stencil

    double *array;
    double *sum;
    int step = n/THREADS_PER_DIM; 
    int size = (N) * sizeof(double);

    int g_size = (step*step) * sizeof(double); 
    array =(double *)malloc(size);
    sum = (double *)malloc(g_size);

    for(int i =0; i<n;i++){
        for(int j =0; j<n; j++){
            array[i*n+j] = pow(1+cos(2*i)+sin(j),2);
        }
    }
    for(int i=0; i<step; i++){
        for(int j=0; j<step; j++){
            sum[i*step+j] =0.0;
        }
    }


    //verify initialization results
    double verisum_1 = verisum_all(n, array);
    double half_value_1 = value_half(n, array);
    double spec_1  = value_37_47(n, array);
    
    //print result
    printf("init verisum all %f\n", verisum_1);
    printf("init verification n/2 %f\n", half_value_1);
    printf("init verification A[37][47] %f\n", spec_1);

    double *dA;
    double *prev_dA;
    double *g_out;
    
    // allocate memory on device
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&prev_dA, size);
    hipMalloc((void **)&g_out, g_size);

    // Copy inputs to device
    hipMemcpy(dA, array, size, hipMemcpyHostToDevice);
    hipMemcpy(prev_dA, array, size, hipMemcpyHostToDevice);
    hipMemcpy(g_out, sum, g_size, hipMemcpyHostToDevice);

    //launch kernal on device
    int t  = 10;
    dim3 dimBlock(THREADS_PER_DIM, THREADS_PER_DIM);
    dim3 dimGrid(n/THREADS_PER_DIM, n/ THREADS_PER_DIM);
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // double v1 =0.0; 
    

    hipEventRecord(start, 0);

    for(int episode =0; episode<t; episode++){
        // printf("loop %d\n", episode );
        calc<<<dimGrid, dimBlock>>>(n, dA, prev_dA);
        hipDeviceSynchronize();

        double *tem_a = dA;
        dA = prev_dA;
        prev_dA = tem_a;  
    }

    reduce<<<dimGrid,dimBlock, dimBlock.x *dimBlock.y *sizeof(double)>>>(prev_dA,n,step, g_out);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    
    hipMemcpy(array,prev_dA, size, hipMemcpyDeviceToHost);
    hipMemcpy(sum,g_out, g_size, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&time, start, stop);
    double verisum=0;
    for(int i=0; i<step*step; i++){
        verisum += sum[i];
    }

    int fl = floor((double)n/2);
    double v2 = array[fl*n+fl];
    double v3 = array[37*n+47];
        //print result
    printf ("Time for the kernel: %f ms\n", time);
    printf("verisum all %f\n", verisum);
    printf("verification n/2 %f\n", v2);
    printf("verification A[37][47] %f\n", v3);


    //free memory
    free(array);
    free(sum);
    hipFree(dA);
    hipFree(prev_dA);
    hipFree(g_out);

    return 0;
}
