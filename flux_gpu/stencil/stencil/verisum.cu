#include "hip/hip_runtime.h"
//
//  stencil main program
//  using CUDA to do parallel computing of stencil
//  do t repeated run
//  do it for n = 500, 1000, 20000 and  t = 10
//  Created by Ethan Zhang on 11/8/18.
//  Copyright © 2018 Ethan Zhang. All rights reserved.
//


#include <iostream>
#include <algorithm>
#include "stdio.h"
#include "cmath"
#include <float.h>
#include "hip/hip_runtime.h"
#include <ctime>

#define THREADS_PER_DIM 25
// #define TASKS_PER_THREADS 50
// #define BLOCKS 32
// #define N 1000*1000
// #define RADIUS  1001
// #define TASKS 
using namespace std;

__device__
double get2ndMin(double *candidates){
    double first, second;
    first = second = DBL_MAX;
    for(int k =0; k<4; k++){
        if(candidates[k]<=first){
            second = first;
            first = candidates[k];
        }
        else if (candidates[k] <= second && candidates[k] >= first){
            second = candidates[k];}
    }
    return second;
}


__global__ 
void calc(int n, double *dA, double *prev_dA){

    int j = threadIdx.y + blockIdx.y * blockDim.y; 
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(i ==0 || i ==n-1 || j ==0 || j ==n-1){
        dA[i*n+j] = prev_dA[i*n+j];
    }else{
        // tmp[lindex_x-1][lindex_y-1] = A[i-1][j-1]
        double candidates[] = {prev_dA[(i+1)*n+(j+1)], prev_dA[(i+1)*n+(j-1)],prev_dA[(i-1)*n+(j-1)],prev_dA[(i-1)*n+(j+1)]};
        dA[i*n+j] = prev_dA[i*n+j] + get2ndMin(candidates);
    }
    __syncthreads();
    // printf("exec. in block%d, threads%d, i%d, j%d, \n", blockIdx.x, threadIdx.x, i, j);
}

//parent node
// __global__ void stencil(double *dA,int n){

//     calc<<<BLOCKS, THREADS_PER_DIM>>>(n, dA); 
//     __syncthreads();
//     printf("exec. in parent node\n");
// }

__global__ void reduce(double *g_idata, double *g_odata) {
    extern __shared__ double sdata[];
    // each thread loads one element from global to shared mem
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}





__global__
void verification(double *A, int n){
    int j = threadIdx.y + blockIdx.y * blockDim.y; 
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    sum += A[i*n+j];
    __syncthreads();

    // A[0] = sum;
    double v2, v3; 
    int fl = floor((double)n/2);
    v2 = A[fl*n+fl];
    v3 = A[37*n+47];
    // __syncthreads()
    A[1] = v2;
    A[2] = v3; 
}


double verisum_all(int n, double *A){
    double sum=0.0;
    for(int i = 0; i<n; i++){
        for(int j=0; j<n; j++){
            sum += A[i*n+j];
        }
    }
    return sum;
}

double value_half(int n, double *A){
    int fl = floor((double)n/2);
    double result  = A[fl*n+ fl];
    return result;
}

double value_37_47(int n, double *A){
    double result =A[37*n+47];
    return result;
}

int main(int argc, char** argv) {
    // initialize below
    int n = atoi(argv[1]);

    int N  = n*n;
    printf("size N%d\n",N);

    // initialize below
    // 1d stencil

    double *array;
    double *sum;
    // int step = n/THREADS_PER_DIM; 
    int size = (N) * sizeof(double);

    int g_size = (step*step) * sizeof(double); 
    array =(double *)malloc(size);
    // sum = (double *)malloc(step);

    for(int i =0; i<n;i++){
        for(int j =0; j<n; j++){
            array[i*n+j] = pow(1+cos(2*i)+sin(j),2);
        }
    }
<<<<<<< HEAD
    for(int i=0; i<step; i++){
        for(int j=0; j<step; j++){
            sum[i*step+j] =0.0;
        }
    }
=======
    // for(int i=0; i<step; i++){
    //     for(int j=0; j<step; j++){
    //         sum[i*n+j] =0.0;
    //     }
    // }
>>>>>>> 7ef1dde9cf28d66ad364f880453a33fe379222b7


    //verify initialization results
    double verisum_1 = verisum_all(n, array);
    double half_value_1 = value_half(n, array);
    double spec_1  = value_37_47(n, array);
    
    //print result
    printf("init verisum all %f\n", verisum_1);
    printf("init verification n/2 %f\n", half_value_1);
    printf("init verification A[37][47] %f\n", spec_1);

    double *dA;
    double *prev_dA;
    // double *g_out;
    
    // allocate memory on device
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&prev_dA, size);
    // hipMalloc((void **)&g_out, size);

    // Copy inputs to device
    hipMemcpy(dA, array, size, hipMemcpyHostToDevice);
    hipMemcpy(prev_dA, array, size, hipMemcpyHostToDevice);
    // hipMemcpy(g_out, sum, size, hipMemcpyHostToDevice);

    //launch kernal on device
    int t  = 10;
    dim3 dimBlock(THREADS_PER_DIM, THREADS_PER_DIM);
    dim3 dimGrid(n/THREADS_PER_DIM, n/ THREADS_PER_DIM);
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // double v1 =0.0; 
    
    hipEventRecord(start, 0);

    for(int episode =0; episode<t; episode++){
        // printf("loop %d\n", episode );
        calc<<<dimGrid, dimBlock>>>(n, dA, prev_dA);
        hipDeviceSynchronize();

        double *tem_a = dA;
        dA = prev_dA;
        prev_dA = tem_a;  
    }
    
    // reduce<<<dimGrid,dimBlock, dimBlock.x *dimBlock.y *sizeof(double)>>>(prev_dA,g_out); //better verification
    // hipMemcpy(sum,g_out, size, hipMemcpyDeviceToHost);
    // double verisum=0;
    // for(int i=0; i<step*step; i++){
    //     verisum += sum[i];
    // }
    
    verification<<<1,1>>>(prev_dA,n); //  verification 
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipMemcpy(array,prev_dA, size, hipMemcpyDeviceToHost);
    
    hipEventElapsedTime(&time, start, stop);
    
        //print result
    printf ("Time for the kernel: %f ms\n", time);
    printf("verisum all %f\n", array[0]);
    printf("verification n/2 %f\n", array[1]);
    printf("verification A[37][47] %f\n", array[2]);





    //free memory
    free(array);
    free(sum);
    hipFree(dA);
    hipFree(prev_dA);
    hipFree(g_out);

    return 0;
}
